#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Authors: Martin Noboa - A01704052
// 		   Bernardo Estrada - A01704320
// Description: This file contains the code to count the number of
//				even numbers within an array using CUDA.
//
// =================================================================
// ======Outputs====================================================
// Single Thread
// sum = 941896832
// avg time = 34.2 ms
//
// CUDA
// sum = 941896832
// avg time = 0.00350 ms
//
// Speedup = 9,771.43x

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void evenNumbers(int *array,int * results){
    __shared__ int cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;
	int acum = 0;
	while (tid < SIZE) {
		if((array[tid]%2)==0){
			acum +=1;
		}
        tid += blockDim.x * gridDim.x;
	}


	cache[cacheIndex] = acum;
    //printf("%i\n", acum);
	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		results[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	int i, *a, *results;
    int *d_a, *d_results;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
    results = (int *) malloc(sizeof(int) * BLOCKS);

    hipMalloc( (void**) &d_a, sizeof(int) * SIZE);
    hipMalloc( (void**) &d_results, sizeof(int) * BLOCKS);

	fill_array(a, SIZE);
    hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	display_array("a", a);

	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();
		
		evenNumbers<<<BLOCKS, THREADS>>>(d_a, d_results);
		
		ms += stop_timer();
	}
    hipMemcpy(results, d_results, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
    int result = 0;
	for (i = 0; i < BLOCKS; i++) {
		result += results[i];
	}
	printf("result = %i\n", result);
	printf("avg time = %.5lf ms\n", (ms / N));
	// must display: result = 500000000

    hipFree(d_a);
    hipFree(d_results);
	free(a);
    free(results);
	return 0;
}
