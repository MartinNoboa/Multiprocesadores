#include "hip/hip_runtime.h"
// =================================================================
//
// File: exercise02.cpp
// Author(s):
//          Martin Adrian Noboa Monar - A01704052
//          Samuel Octavio González Azpeitia - A01704696
//
//			Sin Threads		Con Threads		Speed Up
//			622.582  ms		
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include "utils.h"
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define MAXIMUM 1000001 //1e6
#define THREADS 256
#define BLOCKS	MMIN(32, ((MAXIMUM / THREADS) + 1))
// implement your code

bool isPrime(int number) const{
        bool flag = false;
        if (number < 2) {
            return false;
        }
        for (int i = 2; i <= sqrt(number); i++) {
            if (number % i == 0) {
                return false;
            }
        }
        return true;
}

__global__ void prime(){
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    double result = 0;
	while (tid < MAXIMUM) {
        if (isPrime(tid)){
            result+= tid;
        }
        tid += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	double result;
    double * d_result;
	// These variables are used to keep track of the execution time.
	high_resolution_clock::time_point start, end;
	double timeElapsed;

	cout << "Starting...\n";
	timeElapsed = 0;
	for (int j = 0; j < N; j++) {
		start = high_resolution_clock::now();

		// call the implemented function
        prime<<<BLOCKS, THREADS>>> ();

		end = high_resolution_clock::now();
		timeElapsed += 
			duration<double, std::milli>(end - start).count();
	}
	cout << "result = " << result << "\n";
	cout << "avg time = " << fixed << setprecision(3) 
		 << (timeElapsed / N) <<  " ms\n";

	return 0;
}
