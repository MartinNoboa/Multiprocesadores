#include "hip/hip_runtime.h"
// =================================================================
//
// File: exercise01.cu
// Author(s):
//      Martin Adrian Noboa Monar - A01704052
//      Samuel Octavio González Azpeitia - A01704696
//
//			Sin Threads		Con Threads		Speed Up
//			636.083  ms		
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include "utils.h"
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

// array size
#define SIZE 1000000000
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))
// implement your code
__global__ void even(int *array,int * results){
    __shared__ int cache[THREADS];

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int cacheIndex = threadIdx.x;

    int aux = INT_MAX;
    while (tid < SIZE) {
        if((array[tid]%2)==0){
			aux += array[tid];
		}
        tid += blockDim.x * gridDim.x;
    }

	cache[cacheIndex] = aux;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex+1];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        results[blockIdx.x] = cache[cacheIndex];
    }
}

int main(int argc, char* argv[]) {
    int i, *a, *results;
    int *d_a, *d_r;
    
    // These variables are used to keep track of the execution time.
	high_resolution_clock::time_point start, end;
	double timeElapsed;

	a =  new int[SIZE];
	random_array(a, SIZE);
	display_array("a", a);

	results = new int[BLOCKS];

	hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(int) );

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

	cout << "Starting...\n";
    timeElapsed = 0;
	for (int j = 0; j < N; j++) {
		start = high_resolution_clock::now();

		// call the implemented function
        even<<<BLOCKS, THREADS>>> (d_a, d_r);

		end = high_resolution_clock::now();
		timeElapsed += 
			duration<double, std::milli>(end - start).count();
	}
	hipMemcpy(results, d_r, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

    double aux = 0;
    for (i = 0; i < BLOCKS; i++) {
        aux += results[i];
    }

    cout << "result = " << aux << "\n";
    cout << "avg time = " << fixed << setprecision(3) 
        << (timeElapsed / N) <<  " ms\n";

    hipFree(d_r);
    hipFree(d_a);

    delete [] a;
    delete [] results;

    return 0;
}
