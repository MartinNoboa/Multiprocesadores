// =================================================================
// Multiprocesadores
// File: Exercise03.cpp
// Author: Martin Noboa - A01704052
// 		   Samuel Gonzalez - A01704696
//
//			Sin Threads		Con Threads		Speed Up
//			636.750  ms		281.456  ms		2.26
// =================================================================

#include <iostream>
#include <iomanip>
#include <chrono>
#include "utils.h"
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define SIZE 10000

// implement your code

int main(int argc, char* argv[]) {
	int *array, result;
	// These variables are used to keep track of the execution time.
	high_resolution_clock::time_point start, end;
	double timeElapsed;

	array = new int[SIZE];
	random_array(array, SIZE);
	display_array("before", array);

	cout << "Starting...\n";
	timeElapsed = 0;
	for (int j = 0; j < N; j++) {
		start = high_resolution_clock::now();

		// call the implemented function

		end = high_resolution_clock::now();
		timeElapsed += 
			duration<double, std::milli>(end - start).count();
	}
	display_array("after", array);
	cout << "avg time = " << fixed << setprecision(3) 
		 << (timeElapsed / N) <<  " ms\n";

	delete [] array;
	return 0;
}
