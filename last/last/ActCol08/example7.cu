#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Authors: Martin Noboa - A01704052
// 		   Bernardo Estrada - A01704320
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================
// ======Outputs====================================================
// Single Thread
// sum = 3.7550402023E10
// avg time = 133.3 ms
//
// CUDA
// sum = 3.7550402023E10
// avg time = 0.00230 ms
//
// Speedup = 57,956.52x

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define MAXIMUM 1000000 //1e6
#define THREADS 256
#define BLOCKS	MMIN(32, ((MAXIMUM / THREADS) + 1))


__global__ void prime(int *array){
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int j=2;
    int flag=0;
	while (tid < MAXIMUM) {
            j=2;
            while((j*j)<=tid){
                if(tid%j==0){
                    flag=1;
                    break;
                }
                j++;
            }
            if(flag==0){
                array[tid]=1;
            }
        tid += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	int i, *a;
	double ms;
    int * d_a;

	a = (int *) malloc(sizeof(int) * (MAXIMUM + 1));
	printf("At first, neither is a prime. We will display to TOP_VALUE:\n");
	for (i = 2; i < TOP_VALUE; i++) {
		if (a[i] == 0) {
			printf("%i ", i);
		}
	}
	printf("\n");

    hipMalloc( (void**) &d_a, sizeof(int) * MAXIMUM);
    //hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();

		//primeN(a, MAXIMUM);
        prime<<<BLOCKS, THREADS>>>(d_a);

		ms += stop_timer();
	}

    hipMemcpy(a,d_a, sizeof(int) * MAXIMUM, hipMemcpyDeviceToHost);
	printf("Expanding the numbers that are prime to TOP_VALUE:\n");
	for (i = 2; i < TOP_VALUE; i++) {
		if (a[i] == 1) {
			printf("%i ", i);
		}
	}
	printf("\n");
	printf("avg time = %.5lf ms\n", (ms / N));

	free(a);
	return 0;
}
