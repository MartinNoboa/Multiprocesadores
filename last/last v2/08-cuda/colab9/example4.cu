#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Author(s):
// Description: This file contains the code to count the number of
//				even numbers within an array using CUDA.
//
// =================================================================

/*----------------------------------------------------------------

*

* Multiprocesadores: CUDA

* Fecha: 14-Nov-2021



* Autor: A01209400 - Royer Donnet Arenas Camacho
		 A01654856 - Hugo David Franco Ávila 

*

--------------------------------------------------------------*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void evenNumbers(int *array,int * results){
    __shared__ int cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;
	int acum = 0;
	while (tid < SIZE) {
		if((array[tid]%2)==0){
			acum +=1;
		}
        tid += blockDim.x * gridDim.x;
	}


	cache[cacheIndex] = acum;
    //printf("%i\n", acum);
	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		results[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	int i, *a, *results;
    int *d_a, *d_results;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
    results = (int *) malloc(sizeof(int) * BLOCKS);

    hipMalloc( (void**) &d_a, sizeof(int) * SIZE);
    hipMalloc( (void**) &d_results, sizeof(int) * BLOCKS);

	fill_array(a, SIZE);
    hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	display_array("a", a);

	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		start_timer();
		
		evenNumbers<<<BLOCKS, THREADS>>>(d_a, d_results);
		
		ms += stop_timer();
	}
    hipMemcpy(results, d_results, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);
    int result = 0;
	for (i = 0; i < BLOCKS; i++) {
		result += results[i];
	}
	printf("result = %i\n", result);
	printf("avg time = %.5lf ms\n", (ms / N));
	// must display: result = 500000000

    hipFree(d_a);
    hipFree(d_results);
	free(a);
    free(results);
	return 0;
}