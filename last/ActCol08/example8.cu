#include "hip/hip_runtime.h"
// =================================================================
//
// File: example8.cu
// Authors: Martin Noboa - A01704052
// 		   Bernardo Estrada - A01704320
// Description: This file contains the code that implements the
//				enumeration sort algorithm using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================
// ======Outputs====================================================
// Single Thread
// avg time = 231.18360 ms
//
// CUDA
// avg time = 0.00240 ms
//
// Speedup = 96,325x

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.h"


#define SIZE 10000
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void nSort(int *a,int *b){
	int j;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    while(tid<SIZE){
		for(j=0;j<SIZE;j++){
			if(a[tid]>a[j]||a[tid]==a[j]&&j<tid){
				b[tid]+=1;
			}
		}
		tid += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	int i, *a,*b,*c,*d_a,*d_b;
	double ms;

	a = (int*) malloc(sizeof(int) * SIZE);
	b = (int*) calloc(SIZE,sizeof(int));
	c = (int*) malloc(sizeof(int) * SIZE);
	hipMalloc( (void**) &d_a, sizeof(int) * SIZE);
	hipMalloc( (void**) &d_b, sizeof(int) * SIZE);

	random_array(a, SIZE);
	display_array("before", a);

	hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	printf("Starting...\n");
	ms = 0;
	for (i = 0; i < N; i++) {
		hipMemcpy(d_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);
		start_timer();

		nSort<<<BLOCKS, THREADS>>>(d_a,d_b);

		ms += stop_timer();
	}
	hipMemcpy(b,d_b, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

	for(int i=0;i<SIZE;i++){
		c[b[i]]=a[i];
	}

	for(int i=0;i<SIZE;i++){
		a[i]=c[i];
	}

	display_array("after", a);
	printf("avg time = %.5lf ms\n", (ms / N));

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}

